#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

int n, w;
float p;
float *A, *dist;

void makeAdjency();
void warshallFloyd();
void print_tables(float *table);
void error_check(float *dist, float *cdist);
void cwf1();
void cwf2();
void cwf3();
__global__ void cuda1(int n, int k, float *cA);
__global__ void cuda2(int n, int k, float *cA);
__global__ void cuda3(int n, int k, float *cA, int elements);

int main(int argc, char** argv)
{
	n = 1<<atoi(argv[1]);
	p = atof(argv[2]);
	w = atoi(argv[3]);
    time_t t;
    
    srand((unsigned)time(&t));
	
    A = (float *) malloc (n*n*sizeof(float));
    dist = (float *) malloc (n*n*sizeof(float));

    makeAdjency();
    warshallFloyd();
    cwf1();
    cwf2();
    cwf3();
    
    printf("\n");
    free(A);
    free(dist);
    
    return 0;
}

void makeAdjency() {
    int i, j;
	double r;

    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            A[i*n + j] = 0;
            dist[i*n + j] =0;
        }
    }

    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
			r = (((double)rand()+1)/((double)RAND_MAX+1));
            if (r > p) {
                A[i*n + j] = INFINITY;
                dist[i*n + j] = A[i*n + j];
            }
            else {
                A[i*n + j] = r * w;
                dist[i*n + j] = A[i*n + j];
            }
        }
        A[i*n + i] = 0;
        dist[i*n + i] = 0;
    }

}

void warshallFloyd() {
    int i, j, k;
    double seq_time;
    struct timeval startwtime, endwtime;

    gettimeofday (&startwtime, NULL);
    for (k=0; k<n; k++) {
        for (i=0; i<n; i++) {
            for (j=0; j<n; j++) {
                if ( dist[i*n + j] > dist[i*n + k] + dist[k*n + j] )
                    dist[i*n + j] = dist[i*n + k] + dist[k*n + j];
            }
        }
    }

    gettimeofday (&endwtime, NULL);
    seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
    printf("\n---- n = %d  p = %.2f  w = %d ----\n",n,p,w);
    printf("\nSerial complete time = %f s\n",seq_time);
    
}

__global__ void cuda1(int n, int k, float *cA) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	if ( cA[i*n + j] > cA[i*n + k] + cA[k*n + j] ) {
		cA[i*n + j] = cA[i*n + k] + cA[k*n + j];
	}
    
}

__global__ void cuda2(int n, int k, float *cA) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	extern __shared__ float s[];
	float *ci = s;
	ci[threadIdx.x] = cA[i*n + k];
	
	if ( cA[i*n + j] > ci[threadIdx.x] + cA[k*n + j] ) {
		cA[i*n + j] = ci[threadIdx.x] + cA[k*n + j];
	}
	
}

__global__ void cuda3(int n, int k, float *cA, int elements) {
	int l, m;
	int p = 0, r = 0;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.x + threadIdx.y;
	
	extern __shared__ float s[];
	float *ci = s;
	float *ck = (float *)&ci[blockDim.x*elements];
	
	for (l=i*elements; l<(i*elements) + elements; l++) {
		ci[threadIdx.x*elements + p] = cA[l*n + k];
		p++;
	}
	p = 0;
	
	for (m=j*elements; m<(j*elements) + elements; m++) {
		ck[threadIdx.y*elements + r] = cA[k*n + m];
		r++;
	}
	r = 0;
	
	__syncthreads();
	
	for (l=i*elements; l<(i*elements) + elements; l++) {
		for (m=j*elements; m<(j*elements) + elements; m++) {
			if ( cA[l*n + m] > ci[threadIdx.x*elements + p] + ck[threadIdx.y*elements + r] ) {
				cA[l*n + m] = ci[threadIdx.x*elements + p] + ck[threadIdx.y*elements + r];
			}
			r++;
		}
		r = 0;
		p++;
	}
	
}

void cwf1() {
	float *cA, milliseconds;
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float *dist1 = (float *) malloc (n*n*sizeof(float));
    
    hipMalloc( (void **) &cA, n*n*sizeof(float) );
	hipEventRecord(start);
    hipMemcpy(cA, A, n*n*sizeof(float), hipMemcpyHostToDevice);

	if ( n <= 8 ) {
    	dim3 threadsPerBlock(n, n);
    	int numBlocks = 1;
    	
    	for (int k=0; k<n; k++) {
    		cuda1<<<numBlocks,threadsPerBlock>>>(n,k,cA);
    	}
    }
    else {
    	dim3 threadsPerBlock(8, 8); 
    	dim3 numBlocks(n/threadsPerBlock.x, n/threadsPerBlock.y);
    	
    	for (int k=0; k<n; k++) {
    		cuda1<<<numBlocks,threadsPerBlock>>>(n,k,cA);
    	}
	}
    
    hipMemcpy(dist1, cA, n*n*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("\nCUDA1 complete time = %f s\n",milliseconds/1000);
    
    error_check(dist,dist1);
    
    hipFree(cA);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(dist1);
}

void cwf2() {
	float *cA, milliseconds;
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float *dist2 = (float *) malloc (n*n*sizeof(float));
    
    hipMalloc( (void **) &cA, n*n*sizeof(float) );
	hipEventRecord(start);
    hipMemcpy(cA, A, n*n*sizeof(float), hipMemcpyHostToDevice);

    if ( n <= 8 ) {
    	dim3 threadsPerBlock(n, n);
    	int numBlocks = 1;
    	size_t size = threadsPerBlock.x*sizeof(float);
    	
    	for (int k=0; k<n; k++) {
			cuda2<<<numBlocks,threadsPerBlock,size>>>(n,k,cA);
    	}
    }
    else {
    	dim3 threadsPerBlock(8, 8); 
    	dim3 numBlocks(n/threadsPerBlock.x, n/threadsPerBlock.y);
    	size_t size = threadsPerBlock.x*sizeof(float);
    	
    	for (int k=0; k<n; k++) {
    		cuda2<<<numBlocks,threadsPerBlock,size>>>(n,k,cA);
    	}
	}
    
    hipMemcpy(dist2, cA, n*n*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("\nCUDA2 complete time = %f s\n",milliseconds/1000);
    
    error_check(dist,dist2);
    
    hipFree(cA);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(dist2);
}

void cwf3() {
	int elements = 4;
	float *cA, milliseconds;
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float *dist3 = (float *) malloc (n*n*sizeof(float));
    
    hipMalloc( (void **) &cA, n*n*sizeof(float) );
	hipEventRecord(start);
    hipMemcpy(cA, A, n*n*sizeof(float), hipMemcpyHostToDevice);
    
    if ( n <= (8*elements) ) {
    	dim3 threadsPerBlock(n/elements, n/elements);
    	int numBlocks = 1;
    	size_t size = (threadsPerBlock.x*elements)*sizeof(float) + (threadsPerBlock.y*elements)*sizeof(float);
    	
    	for (int k=0; k<n; k++) {
    		cuda3<<<numBlocks,threadsPerBlock,size>>>(n,k,cA,elements);
    	}
    }
    else {
        dim3 threadsPerBlock(8, 8);
    	dim3 numBlocks(n/(threadsPerBlock.x*elements), n/(threadsPerBlock.y*elements));
    	size_t size = (threadsPerBlock.x*elements)*sizeof(float) + (threadsPerBlock.y*elements)*sizeof(float);
    		
    	for (int k=0; k<n; k++) {
    		cuda3<<<numBlocks,threadsPerBlock,size>>>(n,k,cA,elements);
    	}
	}
    
    hipMemcpy(dist3, cA, n*n*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("\nCUDA3 complete time = %f s\n",milliseconds/1000);
    
    error_check(dist,dist3);
    
    hipFree(cA);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(dist3);
}

void error_check(float *dist, float *cdist) {
	int i;
	int flag = 0;
	
	for (i=0; i<n*n; i++) {
		if (dist[i] != cdist[i]) {
			printf("CUDA Warshall Floyd was failed!\n");
			flag = 1;
			break;
		}
	}
	
	if (flag == 0) printf("CUDA Warshall Floyd was succesful!\n");
	
}

void print_tables(float *table) {
    int i, j;
    
    for (j=0; j<n; j++) {
    	for (i=0; i<n; i++) {
    		printf("%f  ",table[i*n + j]);
    	}
    	printf("\n");
    }

    printf("\n\n");
}

